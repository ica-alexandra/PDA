#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#include <string>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;
// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Cuda convolution function.  Destination will have the same width/height as source
*
* @param source      Source image host
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding x
* @param paddingY    source image padding y
* @param kOffset     offset of kernel data storage in memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination destination image
*/
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, unsigned int kOffset, int kWidth, int kHeight, unsigned char *destination)
{
	// Calculate our pixel's location
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	float sum = 0.0;
	int   pWidth = kWidth / 2;
	int   pHeight = kHeight / 2;

	//Only valid pixels are executed
	if (x >= pWidth + paddingX &&   y >= pHeight + paddingY &&  x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
		y < (blockDim.y * gridDim.y) - pHeight - paddingY)
	{
		for (int j = -pHeight; j <= pHeight; j++)
		{
			for (int i = -pWidth; i <= pWidth; i++)
			{
				// Sample the weight for this location
				int ki = (i + pWidth);
				int kj = (j + pHeight);
				float w = convolutionKernelStore[(kj * kWidth) + ki + kOffset];


				sum += w * float(source[((y + j) * width) + (x + i)]);
			}
		}
	}

	// Medium sum
	destination[(y * width) + x] = (unsigned char)sum;
}

__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	float af = float(a[idx]);
	float bf = float(b[idx]);

	c[idx] = (unsigned char)sqrtf(af*af + bf*bf);
}

// create image buffer
unsigned char* createImageBuffer(unsigned int bytes, unsigned char **devicePtr)
{
	unsigned char *ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
	hipHostGetDevicePointer(devicePtr, ptr, 0);
	return ptr;
}


int main(int argc, char** argv) {
	// open webcam
	cv::VideoCapture camera(0);
	cv::Mat          frame;
	if (!camera.isOpened())
		return -1;

	// capture windows
	//cv::namedWindow("Source");
	//cv::namedWindow("Greyscale");
	//cv::namedWindow("Blurred");
	cv::namedWindow("Sobel");

	// Functions for execution time 
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Create gaussian  kernel(sum = 159)
	const float gaussianKernel5x5[25] =
	{
		2.f / 159.f,  4.f / 159.f,  5.f / 159.f,  4.f / 159.f, 2.f / 159.f,
		4.f / 159.f,  9.f / 159.f, 12.f / 159.f,  9.f / 159.f, 4.f / 159.f,
		5.f / 159.f, 12.f / 159.f, 15.f / 159.f, 12.f / 159.f, 5.f / 159.f,
		4.f / 159.f,  9.f / 159.f, 12.f / 159.f,  9.f / 159.f, 4.f / 159.f,
		2.f / 159.f,  4.f / 159.f,  5.f / 159.f,  4.f / 159.f, 2.f / 159.f,
	};
	hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), gaussianKernel5x5, sizeof(gaussianKernel5x5), 0);
	const unsigned int gaussianKernel5x5Offset = 0;

	// Sobel gradient kernels
	const float sobelGradientX[9] =
	{
		-1.f, 0.f, 1.f,
		-2.f, 0.f, 2.f,
		-1.f, 0.f, 1.f,
	};
	const float sobelGradientY[9] =
	{
		1.f, 2.f, 1.f,
		0.f, 0.f, 0.f,
		-1.f, -2.f, -1.f,
	};
	hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientX, sizeof(sobelGradientX), sizeof(gaussianKernel5x5));
	hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientY, sizeof(sobelGradientY), sizeof(gaussianKernel5x5) + sizeof(sobelGradientX));
	const unsigned int sobelGradientXOffset = sizeof(gaussianKernel5x5) / sizeof(float);
	const unsigned int sobelGradientYOffset = sizeof(sobelGradientX) / sizeof(float) + sobelGradientXOffset;

	// Create CPU/GPU shared images
	camera >> frame;
	unsigned char *sourceDataDevice, *blurredDataDevice, *edgesDataDevice;
	cv::Mat source(frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &sourceDataDevice));
	cv::Mat blurred(frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &blurredDataDevice));
	cv::Mat edges(frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &edgesDataDevice));

	// Create 2 temporary images(sobel gradients)
	unsigned char *deviceGradientX, *deviceGradientY;
	hipMalloc(&deviceGradientX, frame.size().width * frame.size().height);
	hipMalloc(&deviceGradientY, frame.size().width * frame.size().height);

	//capture images
	while (1)
	{
		// grey scale picture
		camera >> frame;
		cvtColor(frame, source, CV_BGR2GRAY);

		// record start time
		hipEventRecord(start);
		{
			// convolution kernel  parameters
			dim3 cblocks(frame.size().width / 16, frame.size().height / 16);
			dim3 cthreads(16, 16);

			// pythagoran kernel parameters
			dim3 pblocks(frame.size().width * frame.size().height / 256);
			dim3 pthreads(256, 1);

			//  gaussian blur (first kernel in store at 0)
			convolve << <cblocks, cthreads >> >(sourceDataDevice, frame.size().width, frame.size().height, 0, 0, gaussianKernel5x5Offset, 5, 5, blurredDataDevice);

			// sobel gradient convolutions (x&y padding is now 2 because there is a border of 2 around a 5x5 gaussian filtered image)
			convolve << <cblocks, cthreads >> >(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientXOffset, 3, 3, deviceGradientX);
			convolve << <cblocks, cthreads >> >(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientYOffset, 3, 3, deviceGradientY);
			pythagoras << <pblocks, pthreads >> >(deviceGradientX, deviceGradientY, edgesDataDevice);

			hipDeviceSynchronize();
		}
		hipEventRecord(stop);

		// show execution time
		float ms = 0.0f;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		std::cout << "Elapsed GPU time: " << ms << " milliseconds" << std::endl;

		// Muestra resultados
		//imshow("Source", frame);
		//imshow("Greyscale", source);
		//imshow("Blurred", blurred);
		imshow("Sobel", edges);

		// Spin
		if (cv::waitKey(1) == 27) break;
	}

	// Exit
	hipHostFree(source.data);
	hipHostFree(blurred.data);
	hipHostFree(edges.data);
	hipFree(deviceGradientX);
	hipFree(deviceGradientY);

	return 0;
}